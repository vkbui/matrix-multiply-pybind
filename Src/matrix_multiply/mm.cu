#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples
#include <hip/hip_runtime_api.h>       // helper functions for CUDA error checking and initialization

#define BDIMX 16
#define BDIMY 16
#define NUM_STREAMS 4  

// shared mem version for better optimization
__global__ void matrixMultiplication(float *C, float *A, float *B, const int A_rows, const int A_cols, const int B_cols)
{
    // allocate shared memory for tile
    __shared__ float A_shared[BDIMY][BDIMX+1]; // padding for less bank conflicts
    __shared__ float B_shared[BDIMX][BDIMX+1];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockIdx.y * blockDim.y + ty;
    int col = blockIdx.x * blockDim.x + tx;
    
    float sum = 0.0f;
    
    // loop over the tiles of A and B
    for (int t = 0; t < (A_cols + BDIMX - 1) / BDIMX; t++) {
        // load tiles into shared memory
        if (row < A_rows && t * BDIMX + tx < A_cols) {
            A_shared[ty][tx] = A[row * A_cols + t * BDIMX + tx];
        } 
        else {
            A_shared[ty][tx] = 0.0f;
        }
        
        if (t * BDIMX + ty < A_cols && col < B_cols) {
            B_shared[ty][tx] = B[(t * BDIMX + ty) * B_cols + col];
        } 
        else {
            B_shared[ty][tx] = 0.0f;
        }
        
        __syncthreads(); // ensure all threads have finished loading data in shared mem
        
        // compute partial sum for this tile
        for (int k = 0; k < BDIMX; k++) {
            sum += A_shared[ty][k] * B_shared[k][tx];
        }
        
        __syncthreads(); // ensure all threads have finished using current data
    }
    
    if (row < A_rows && col < B_cols) {
        C[row * B_cols + col] = sum;
    }
}

#define INDEX(ROW, COL, INNER) ((ROW) * (INNER) + (COL))

void initialData(float *in, const int size)
{
    for (int i = 0; i < size; i++)
    {
        in[i] = (float)(rand() & 0xFF) / 10.0f;
    }

    return;
}

void checkResult(float *hostRef, float *gpuRef, int rows, int cols)
{
    double epsilon = 1.0E-1;
    bool match = 1;

    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            int index = INDEX(i, j, cols);
            if (abs(hostRef[index] - gpuRef[index]) > epsilon) {
                match = 0;
                printf("different on (%d, %d) (offset=%d) element in "
                        "matrix: host %f gpu %f\n", i, j, index,
                        hostRef[index], gpuRef[index]);
                break;
            }
        }
        if (!match) break;
    }

    if (match)
        printf("PASS\n\n");
    else
        printf("FAIL\n\n");
}

void matrixMultiplicationHost(float *C, float *A, float *B, const int A_rows, const int A_cols, const int B_cols)
{
    for (int i = 0; i < A_rows; i++) {
        for (int j = 0; j < B_cols; j++) {
            float sum = 0.0f;
            for (int k = 0; k < A_cols; k++) {
                sum += A[i * A_cols + k] * B[k * B_cols + j];
            }
            C[i * B_cols + j] = sum;
        }
    }
}

void cu_matrixMultiply(float* A, float* B, float* C, int A_rows, int A_cols, int B_cols)
{

    int B_rows = A_cols;
    
    printf("Matrix A: %d x %d\n", A_rows, A_cols);
    printf("Matrix B: %d x %d\n", B_rows, B_cols);
    printf("Matrix C: %d x %d\n", A_rows, B_cols);

    size_t A_bytes = A_rows * A_cols * sizeof(float);
    size_t B_bytes = B_rows * B_cols * sizeof(float);
    size_t C_bytes = A_rows * B_cols * sizeof(float);

    // calculate per-stream sizes 
    int rows_per_stream = A_rows / NUM_STREAMS;
    size_t A_bytes_per_stream = rows_per_stream * A_cols * sizeof(float);
    size_t C_bytes_per_stream = rows_per_stream * B_cols * sizeof(float);

    //float* h_A, * h_B;

    // pinned memory for streams
    //checkCudaErrors(hipHostAlloc((void**)&h_A, A_bytes, hipHostMallocDefault));
    //checkCudaErrors(hipHostAlloc((void**)&h_B, B_bytes, hipHostMallocDefault));
    //checkCudaErrors(hipHostAlloc((void**)&gpuRef, C_bytes, hipHostMallocDefault));


    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        checkCudaErrors(hipStreamCreate(&streams[i]));
    }

    float* d_A, * d_B, * d_C;
    checkCudaErrors(hipMalloc((float**)&d_A, A_bytes));
    checkCudaErrors(hipMalloc((float**)&d_B, B_bytes));
    checkCudaErrors(hipMalloc((float**)&d_C, C_bytes));

    // copy matrix B to device since used by all streams
    checkCudaErrors(hipMemcpy(d_B, B, B_bytes, hipMemcpyHostToDevice));

    dim3 block(BDIMX, BDIMY);
    dim3 grid_segment((B_cols + block.x - 1) / block.x, (A_rows + block.y - 1) / block.y);

    for (int i = 0; i < NUM_STREAMS; i++) {
        size_t A_offset = i * rows_per_stream * A_cols;
        size_t C_offset = i * rows_per_stream * B_cols;

        // copy segment of A from host to device
        checkCudaErrors(hipMemcpyAsync(&d_A[A_offset], &A[A_offset], A_bytes_per_stream, hipMemcpyHostToDevice, streams[i]));

        // call kernel
        matrixMultiplication <<<grid_segment, block, 0, streams[i] >> > (&d_C[C_offset], &d_A[A_offset], d_B, rows_per_stream, A_cols, B_cols);

        // copy segment of C back to host
        checkCudaErrors(hipMemcpyAsync(&C[C_offset], &d_C[C_offset], C_bytes_per_stream, hipMemcpyDeviceToHost, streams[i]));
    }

    // synchronize streams
    for (int i = 0; i < NUM_STREAMS; i++) {
        checkCudaErrors(hipStreamSynchronize(streams[i]));
    }

    //hipMemcpyFromSymbol(&out, HIP_SYMBOL(d_), C_bytes);
    // Free device memory
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));

    /*checkCudaErrors(hipHostFree(h_A));
    checkCudaErrors(hipHostFree(h_B));
    checkCudaErrors(hipHostFree(gpuRef));
    return out;*/
}

int main(int argc, char **argv)
{
    // matrix dimensions
    int A_rows = 1024;    
    int A_cols = 512;     
    int B_rows = A_cols; // rows and cols need to be same
    int B_cols = 256;     

    printf("Matrix A: %d x %d\n", A_rows, A_cols);
    printf("Matrix B: %d x %d\n", B_rows, B_cols);
    printf("Matrix C: %d x %d\n", A_rows, B_cols);

    size_t A_bytes = A_rows * A_cols * sizeof(float);
    size_t B_bytes = B_rows * B_cols * sizeof(float);
    size_t C_bytes = A_rows * B_cols * sizeof(float);

    // calculate per-stream sizes 
    int rows_per_stream = A_rows / NUM_STREAMS;
    size_t A_bytes_per_stream = rows_per_stream * A_cols * sizeof(float);
    size_t C_bytes_per_stream = rows_per_stream * B_cols * sizeof(float);
    
    float *h_A, *h_B, *hostRef, *gpuRef;
    
    // pinned memory for streams
    checkCudaErrors(hipHostAlloc((void**)&h_A, A_bytes, hipHostMallocDefault));
    checkCudaErrors(hipHostAlloc((void**)&h_B, B_bytes, hipHostMallocDefault));
    checkCudaErrors(hipHostAlloc((void**)&hostRef, C_bytes, hipHostMallocDefault));
    checkCudaErrors(hipHostAlloc((void**)&gpuRef, C_bytes, hipHostMallocDefault));

    // Initialize host arrays
    initialData(h_A, A_rows * A_cols);
    initialData(h_B, B_rows * B_cols);

    // host matrix multiplication
    matrixMultiplicationHost(hostRef, h_A, h_B, A_rows, A_cols, B_cols);

    dim3 block(BDIMX, BDIMY);
    dim3 grid_segment((B_cols + block.x - 1) / block.x, (rows_per_stream + block.y - 1) / block.y); // for streams

    // streams for DMA transfers and computation overlap
    printf("Using %d CUDA streams with DMA transfers\n", NUM_STREAMS);
    printf("Rows per stream: %d\n", rows_per_stream);
    printf("Launching with grid %d x %d and block %d x %d per stream\n", grid_segment.x, grid_segment.y, block.x, block.y);

    // create streams
    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        checkCudaErrors(hipStreamCreate(&streams[i]));
    }

    float *d_A, *d_B, *d_C;
    checkCudaErrors(hipMalloc((float**)&d_A, A_bytes));
    checkCudaErrors(hipMalloc((float**)&d_B, B_bytes));
    checkCudaErrors(hipMalloc((float**)&d_C, C_bytes));

    // copy matrix B to device since used by all streams
    checkCudaErrors(hipMemcpy(d_B, h_B, B_bytes, hipMemcpyHostToDevice));

    for (int i = 0; i < NUM_STREAMS; i++) {
        size_t A_offset = i * rows_per_stream * A_cols;
        size_t C_offset = i * rows_per_stream * B_cols;
        
        // copy segment of A from host to device
        checkCudaErrors(hipMemcpyAsync(&d_A[A_offset], &h_A[A_offset], A_bytes_per_stream, hipMemcpyHostToDevice, streams[i]));
        
        // call kernel
        matrixMultiplication<<<grid_segment, block, 0, streams[i]>>>(&d_C[C_offset], &d_A[A_offset], d_B, rows_per_stream, A_cols, B_cols);

        // copy segment of C back to host
        checkCudaErrors(hipMemcpyAsync(&gpuRef[C_offset], &d_C[C_offset], C_bytes_per_stream, hipMemcpyDeviceToHost, streams[i]));
    }

    // synchronize streams
    for (int i = 0; i < NUM_STREAMS; i++) {
        checkCudaErrors(hipStreamSynchronize(streams[i]));
    }

    // Free device memory
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));

    checkResult(hostRef, gpuRef, A_rows, B_cols);
    printf("Matrix multiplication completed\n");

    checkCudaErrors(hipHostFree(h_A));
    checkCudaErrors(hipHostFree(h_B));
    checkCudaErrors(hipHostFree(hostRef));
    checkCudaErrors(hipHostFree(gpuRef));

    
    return EXIT_SUCCESS;
}